#include "hip/hip_runtime.h"
#include <iostream>
#include <numeric>
#include <math.h>

__constant__ float epsilonSquared = 0.2;
__constant__ float G = 6.67300E-11;
__device__ float globalDt;

const int MAX_THREAD_SIZE = 1024;

__global__ void generateParticles(float4 particle) {

}

__device__ int getGlobalId() {
	return blockIdx.x * blockDim.y * blockDim.x + threadIdx.y * blockDim.x + threadIdx.x;
}

__device__ int getParticleId() {
	return blockIdx.x * blockDim.y + threadIdx.y;
}

inline __device__ float3 segmentFloat4(float4 f4) {
	return make_float3(f4.x, f4.y, f4.z);
}

__device__ void calculatePartitionAcceleration(float3 *acceleration) {
	for (int q = 0; q < blockDim.x; q++) {
		float3 tempA = calculateBodyAcceleration(body, interactingBodies[q]);
		acceleration->x += tempA.x;
		acceleration->y += tempA.y;
		acceleration->z += tempA.z;
	}
}

__device__ float3 calculateBodyAcceleration(float4 bi, float4 bj) {
	float3 rij(bi.x - bj.x, bi.y - bj.y, bi.z - bj.z);
	float3 partialAcc (rij.x * bj.w, rij.y * bj.w, rij.z * bj.z);
	float smoothing = (rij.x * rij.x + rij.y * rij.y + rij.z * rij.z + epsilonSquared);
	smoothing = smoothing * smoothing * smoothing;
	smoothing = sqrtf(smoothing);
	partialAcc.x /= smoothing;
	partialAcc.y /= smoothing;
	partialAcc.z /= smoothing;
	return partialAcc;
}

__device__ float3 updateBodyVelocity(float3 a, float4 v, float dt) { //velocity verlet
	float3 newV;
	newV.x = v.x + 0.5 * a.x * dt;
	newV.y = v.y + 0.5 * a.y * dt;
	newV.z = v.z + 0.5 * a.z * dt;
	return newV;
}

__device__ void updateBodyPosition(float3 velocity, float4 *r, float dt) {
	r->x += acceleration.x * dt;
	r->y += acceleration.y * dt;
	r->z += acceleration.z * dt;
}

__global__ void simulateNaive(float4 *bodies, float3 *dynamics, int n_particles, float _dt, int epochs) {
	const int MAX_THREAD_COUNT = 1024;
	int particleId = blockDim.x * blockIdx.x + threadIdx.x;
	int nParticles = n_particles;
	float dt = _dt;
	float4 body = bodies[particleId];
	float4 dynamic = dynamics[particleId];
	float3 velocity(dynamic.x, dynamic.y, dynamic.z);
	float3 r(body.x, body.y, body.z);
	float3 acceleration(0.0f, 0.0f, 0.0f);
	extern __shared__ float4 interactingBodies[];

	for (int j = 0; j < epochs; j++) {
		float3 vHalf = updateBodyVelocity(acceleration, velocity, dt, true);
		body.x += vHalf.x * dt;
		body.y += vHalf.y * dt;
		body.z += vHalf.z * dt;
		for (int i = 0; i < n_particles; i += blockDim.x) {
			interactingBodies[threadIdx.x] = bodies[threadIdx.x + i];
			__syncthreads();
			calculatePartitionAcceleration(&acceleration);
			__syncthreads();
		}
		acceleration.x *= G;
		acceleration.y *= G;
		acceleration.z *= G;
		velocity = updateBodyVelocity(acceleration, vHalf, dt);
	}
}

void beginUniverseSimulation(int numberOfParticles, int partitions, float dt, int epochs) { //add ability to serialize from past renders.
	size_t allocationSize = sizeof(float4) * numberOfParticles;
	float4 *bodies = malloc(allocationSize);
	float4 *dynamics = malloc(allocationSize);
	float4 *dBodies;
	float4 *dDynamics;
	dim3 blocks(numberOfParticles / partitions, 0, 0);
	dim3 threads(partitions, 0, 0);

	hipMalloc((void**) &dBodies, allocationSize);
	hipMalloc((void**) &dDynamics, allocationSize);
	hipMemcpy(dBodies, bodies, hipMemcpyHostToDevice);
	hipMemcpy(dDynamics, dynamics, hipMemcpyHostToDevice);
	simulateNaive<<<blocks, threads, sizeof(float4) * partitions>>>(d_particles, dDynamics, numberOfParticles, dt, epochs);
}

void resumeUniverseSimulation() {

}
