#include "hip/hip_runtime.h"
#include <iostream>
#include <numeric>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "hip/hip_runtime.h"
#include "UniverseSimulation.cuh"
#include <stdio.h>
#include "../CudaUtilities.cuh"

__constant__ float G = 6.67300E-11;

template<typename F3, typename F4>
__device__ void calculatePartitionAcceleration(F4 body, F3 *acceleration) {
	for (int q = 0; q < blockDim.x; q++) {
		F3 tempA = calculateBodyAcceleration(body, interactingBodies[q]);
		acceleration->x += tempA.x;
		acceleration->y += tempA.y;
		acceleration->z += tempA.z;
	}
}

template<typename F, typename F3, typename F4>
__device__ F3 calculateBodyAcceleration(F4 bi, F4 bj) {
	F3 rij(bi.x - bj.x, bi.y - bj.y, bi.z - bj.z);
	F3 partialAcc (rij.x * bj.w, rij.y * bj.w, rij.z * bj.z);
	F smoothing = (rij.x * rij.x + rij.y * rij.y + rij.z * rij.z + epsilonSquared);
	smoothing = smoothing * smoothing * smoothing;
	smoothing = sqrtf(smoothing);
	partialAcc.x /= smoothing;
	partialAcc.y /= smoothing;
	partialAcc.z /= smoothing;
	return partialAcc;
}

template<typename F, typename F3, typename F4>
__device__ float3 updateBodyVelocity(F3 a, F4 v, F dt) { //velocity verlet
	float3 newV;
	newV.x = v.x + 0.5 * a.x * dt;
	newV.y = v.y + 0.5 * a.y * dt;
	newV.z = v.z + 0.5 * a.z * dt;
	return newV;
}

template<typename F, typename F3, typename F4>
__device__ void updateBodyPosition(F3 velocity, F4 *r, float dt) {
	r->x += acceleration.x * dt;
	r->y += acceleration.y * dt;
	r->z += acceleration.z * dt;
}

template<typename F, typename F3, typename F4>
__global__ void simulateNaive(F4 *bodies, F4 *dynamics, F _dt, F _epsilon, int n_particles) {
	const int MAX_THREAD_COUNT = 1024;
	F dt = _dt;
	int particleId = blockDim.x * blockIdx.x + threadIdx.x;
	int nParticles = n_particles;
	F4 body = bodies[particleId];
	F4 dynamic = dynamics[particleId];
	F3 velocity(dynamic.x, dynamic.y, dynamic.z);
	F3 r(body.x, body.y, body.z);
	F3 acceleration(0.0f, 0.0f, 0.0f);
	extern __shared__ float4 interactingBodies[];

	F3 vHalf = updateBodyVelocity(acceleration, velocity, dt, true);
	body.x += vHalf.x * dt;
	body.y += vHalf.y * dt;
	body.z += vHalf.z * dt;
	for (int i = 0; i < n_particles; i += blockDim.x) {
		interactingBodies[threadIdx.x] = bodies[threadIdx.x + i];
		__syncthreads();
		calculatePartitionAcceleration(&acceleration);
		__syncthreads();
	}
	acceleration.x *= G;
	acceleration.y *= G;
	acceleration.z *= G;
	velocity = updateBodyVelocity(acceleration, vHalf, dt);
	dynamic.x = velocity.x;
	dynamic.y = velocity.y;
	dynamic.z = velocity.z;

	printf("Particle: %i \n", particleId);
	printf("Acceleration: %f %f %f \n", acceleration.x, acceleration.y, acceleration.z);
	printf("Velocity: %f %f %f \n", velocity.x, velocity.y, velocity.z);
	printf("Position: %f %f %f \n", body.x, body.y, body.z);
	printf("\n");

	bodies[particleId] = body;
	dynamics[particleId] = dynamic;
}

template<typename F4>
__global__ void generateParticles(hiprandState *_state, float4 *bodies, float4 *dynamics, float4 *ranges) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	F4 body = bodies[idx];
	F4 dynamic = dynamics[idx];
	F4 range = ranges[idx];
	F4 range2 = ranges[idx + 1];
	hiprandState state = _state[idx];

	body.x = (hiprand_uniform(&state) * (range.x - range.y + 0.999999)) + range.y;
	body.y = (hiprand_uniform(&state) * (range.x - range.y + 0.999999)) + range.y;
	body.z = (hiprand_uniform(&state) * (range.x - range.y + 0.999999)) + range.y;
	dynamic.x = (hiprand_uniform(&state) * (range.z - range.w + 0.999999)) + range.w;
	dynamic.y = (hiprand_uniform(&state) * (range.z - range.w + 0.999999)) + range.w;
	dynamic.z = (hiprand_uniform(&state) * (range.z - range.w + 0.999999)) + range.w;
	body.w = (hiprand_uniform(&state) * (range2.x - range2.y + 0.999999)) + range2.y;
}

template<class F, class F3, class F4>
void beginSimulation(UniverseSimSpec<F> *spec, F4 *ranges) {
	size_t allocationSize = sizeof(F4) * spec->particles;
	F dt = spec->dt;
	F epsilon = spec->epsilon;
	F4 *bodies = (F4 *)malloc(allocationSize);
	F4 *dynamics = (F4 *)malloc(allocationSize);
	F4 *states = (F4 *)malloc(allocationSize);
	F3 *accelerations = (F3 *)malloc(allocationSize);
	F4 *generationRanges = (F4 *)malloc(allocationSize * 2); //[pos_h, pos_l, vel_h, vel_l]; [mass_h, mass_l, etc, etc]
	F4 *dBodies, *dDynamics, *dGenerationRanges;
	F3 *dAccelerations;
	dim3 blocks(spec->particles / spec->partitions, 0, 0);
	dim3 threads(spec->partitions, 0, 0);
	hiprandState *dStates;

	hipMalloc(&dStates, allocationSize);
	hipMemcpy(dStates, states, allocationSize, hipMemcpyHostToDevice);
	cudaAlloCopy<F4>(bodies, dBodies, allocationSize);
	cudaAlloCopy<F4>(dynamics, dDynamics, allocationSize);
	cudaAlloCopy<F4>(ranges, dBodies, allocationSize * 2);
	cudaAlloCopy<F3>(accelerations, dAccelerations, allocationSize);
	generateParticles<F4><<<blocks, threads>>>(dStates, dBodies, dDynamics, dGenerationRanges);

	for (int i = 0; i < spec->epochs; i++) {
		simulateNaive<F, F3, F4><<<blocks, threads, sizeof(F4) * spec->partitions>>>(dBodies, dDynamics, dt, epsilon, spec->particles);
		hipMemcpy(bodies, dBodies, hipMemcpyDeviceToHost); //copy back to save to binary file
		hipMemcpy(dynamics, dDynamics, hipMemcpyDeviceToHost);
		hipMemcpy(generationRanges, dGenerationRanges, hipMemcpyDeviceToHost);
		hipMemcpy(accelerations, dAccelerations, hipMemcpyDeviceToHost); //yo lance sucks
	}
}
template void beginSimulation <float, float3, float4>(UniverseSimSpec<float> *, float4 *);
template void beginSimulation <double, double3, double4>(UniverseSimSpec<double> *, double4 *);
