#include "CudaUtilities.cuh"
#include <typeinfo>

namespace Helix {
	void *fixedCudaMalloc(size_t len) {
		void *p;
		if (hipMalloc(&p, len) == hipSuccess) return p;
		return 0;
	}

	template<typename F>
	F *cudaAlloCopy(F *org, const size_t size) {
		void *mem = fixedCudaMalloc(size);
		F 	 *res = (F *)mem;
		hipMemcpy(res, org, size, hipMemcpyHostToDevice);
		return res;
	}
	template float *cudaAlloCopy <float> (float *, const size_t);
	template double *cudaAlloCopy <double> (double *, const size_t);
}
