#include <typeinfo>
#include "CudaUtilities.cuh"
#include "hip/hip_runtime.h"

namespace Helix {

void *fixedCudaMalloc(size_t len) {
	void *p;
	if (hipMalloc(&p, len) == hipSuccess) return p;
	return 0;
}

bool gpuSupported() {
	int devices = 0;
	hipError_t err = hipGetDeviceCount(&devices);
	return devices > 0 && err == hipSuccess;
}

template<typename F>
F *cudaAlloCopy(F *org, const size_t size) {
	void *mem = fixedCudaMalloc(size);
	F 	 *res = (F *)mem;
	hipMemcpy(res, org, size, hipMemcpyHostToDevice);
	return res;
}
template float *cudaAlloCopy <float> (float *, const size_t);
template double *cudaAlloCopy <double> (double *, const size_t);
}
