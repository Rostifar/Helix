#include "hip/hip_runtime.h"
#include "CudaUtilities.cuh"

void *fixedCudaMalloc(size_t len) {
	void *p;
	if (hipMalloc(&p, len) == hipSuccess) return p;
	return 0;
}

template<typename FN>
void cudaAlloCopy(FN *org, const size_t size) {
	void* mem = fixedCudaMalloc(size);
	FN *res = (FN *)mem;
	hipMemcpy(res, org, size, hipMemcpyHostToDevice);
}
template void cudaAlloCopy <float4>(float4 *, float4 *, const size_t);
template void cudaAlloCopy <float3>(float3 *, float3 *, const size_t);
template void cudaAlloCopy <double4>(double4 *, double4 *, const size_t);
template void cudaAlloCopy <double3>(double3 *, double3 *, const size_t);
